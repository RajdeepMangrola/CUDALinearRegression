/* ---------------------------------------------------------------------------
** Matrix.cu
** See Matrix.h for more details
**
** Author: Vishnu T Suresh
** -------------------------------------------------------------------------*/
#include "Matrix.h"
#include <iostream>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <sstream>

#include "hipblas.h"
#include <cula.h>
#include <cula_lapack.h>
Matrix::Matrix(std::string filename){
	w=0;
	h=0;
	trans=false;
	std::ifstream infile(filename.c_str());
	bool ind_var=false;
	if(infile.is_open()){
		std::string line = "";
		while (getline(infile, line))
		{
			h++;
			std::stringstream strstr(line);
			std::string value = "";
			while (std::getline(strstr,value, ','))
			{
				if(!ind_var)
				{
					w++;
				}
				v.push_back((float)atof(value.c_str()));

			}
			ind_var=true;
		}
	}
	else
		exit(EXIT_FAILURE);
}
float& Matrix::operator()(int i,int j){
	return v[(j-1)*w +(i-1)];
}
std::ostream& operator<< (std::ostream &out, Matrix mat){
	if(!mat.trans)
		for(int j=1;j<=mat.h;j++)
		{
			for(int i=1;i<=mat.w;i++)
			{
				out<<mat(i,j)<<"\t";
			}
			out<<"\n";
		}
	else
		for(int j=1;j<=mat.w;j++)
		{
			for(int i=1;i<=mat.h;i++)
			{
				out<<mat(j,i)<<"\t";
			}
			out<<"\n";
		}
	
	return out;
}
Matrix Matrix::operator* (Matrix mat){
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;

	float* dv,*dv2,*dc;
	cudaStat = hipMalloc ((void**)&dv, w*h*sizeof(float));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		exit(EXIT_FAILURE);
	}
	cudaStat = hipMalloc ((void**)&dv2, (mat.w)*(mat.h)*sizeof(float));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		exit(EXIT_FAILURE);
	}

	int ch,cw,k;
	hipblasOperation_t transa,transb;
	if(trans)
	{
		ch=w;
		k=h;
		transa=HIPBLAS_OP_N;
	}
	else
	{
		ch=h;
		k=w;
		transa=HIPBLAS_OP_T;
	}
	if(mat.trans)
	{
		cw=mat.h;
		transb=HIPBLAS_OP_N;
	}
	else
	{
		cw=mat.w;
		transb=HIPBLAS_OP_T;
	}
	cudaStat = hipMalloc ((void**)&dc, ch*cw*sizeof(float));
	if (cudaStat != hipSuccess) {
		printf ("device memory allocation failed");
		exit(EXIT_FAILURE);
	}
	stat = hipblasCreate(&handle);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("CUBLAS initialization failed\n");
		exit(EXIT_FAILURE);
	}
	stat = hipblasSetMatrix (w, h, sizeof(float), &v[0], w, dv, w);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("data download failed");
		hipFree (dv);
		hipblasDestroy(handle);
		exit(EXIT_FAILURE);
	}
	stat = hipblasSetMatrix (mat.w, mat.h, sizeof(float), &(mat.v[0]), mat.w, dv2, mat.w);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf ("data download failed");
		hipFree (dv);
		hipblasDestroy(handle);
		exit(EXIT_FAILURE);
	}
	const float alpha=1.0f,beta=0.0f;
	hipblasSgemm(handle,transa,transb,ch,cw,k,&alpha,dv,w,dv2,mat.w,&beta,dc,ch);
	Matrix C(ch,cw);
	C.trans=true;
	stat=hipblasGetMatrix(cw,ch,sizeof(float),dc,cw,&(C.v[0]),cw);
	hipFree(dv);
	hipFree(dv2);
	hipFree(dc);
	if (stat != HIPBLAS_STATUS_SUCCESS) {
		if(stat == HIPBLAS_STATUS_INVALID_VALUE)
			fprintf (stderr, "!!!! invalid value\n");
		fprintf (stderr, "!!!! device read error\n");
		exit(EXIT_FAILURE);
	}

	return C;
}
Matrix::Matrix(int width,int height){
	w=width;
	h=height;
	trans=false;
	v.resize(w*h,0);
}
Matrix trans(Matrix mat){
	mat.trans=true;
	return mat;
}
Matrix inverse(Matrix mat){
	int size=mat.h;
	int *ipiv;
	float *cache,*devOutput,*Output;
	int size2 = size * size * sizeof(float);
	
	Output=(float*)malloc(size2);
	hipMalloc(&ipiv, size2);
	hipMalloc(&cache, size2);
	hipMalloc(&devOutput, size2);
	
	hipMemcpy(cache, &(mat.v[0]), size2, hipMemcpyHostToDevice);
	hipMemcpy(devOutput, &((Matrix::IdentityMatrix(size)).v[0]), size2, hipMemcpyHostToDevice);
	
	culaInitialize();
	culaDeviceSgesv(size, size, cache, size, ipiv, devOutput, size);
	culaShutdown();
	hipMemcpy(Output,devOutput, size2, hipMemcpyDeviceToHost);
		for(int j = 1; j <= size; ++j)
		        	for(int i = 1; i <= size; ++i)
		        		mat(i,j)=Output[(j-1)*size+(i-1)];
	hipFree(ipiv);
	hipFree(cache);
	hipFree(devOutput);
	free(Output);
	return mat;
}
Matrix Matrix::IdentityMatrix(int n){
	Matrix mat(n,n);
	for(int i=1;i<=n;i++)
	{
		mat(i,i)=1;
	}
	return mat;
}
int Matrix::height(){
	return h;
}
int Matrix::width(){
	return w;
}
